// gemm_simple3.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "gemm_logic/3_gemm_tiled.cuh"
#include "gemm_samples.cuh"
#include "gemm_utils.h"

//#define TILE_WIDTH 16
// Each || is 16x16     //to refactoring
//
//   || ||     || || || ||     || || || ||
//   || ||  X               =  || || || ||
//   || ||     || || || ||     || || || ||

int main() {
    const gemm::Gemm& data = gemm::complicated_sample;


PROFILE_REPEAT(
    gemm::gemm_tiled_run(data);
);

    std::vector<float> h_C = gemm::gemm_tiled_run(data);

    std::cout << "sample3 gemm tiled, Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), data.M, data.N);

    CudaTimer::printAll();
    return 0;
}
