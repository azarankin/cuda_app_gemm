// gemm_simple2.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "gemm_logic/2_gemm_naive.cuh"
#include "gemm_samples.cuh"
#include "gemm_utils.h"

// sample 2
//
//   || ||     || || || ||     || || || ||
//   || ||  X               =  || || || ||
//   || ||     || || || ||     || || || ||

int main()
{
    const gemm::Gemm& data = gemm::complicated_sample;
    
PROFILE_REPEAT(
    gemm::gemm_naive_run(data);
);
    std::vector<float> h_C = gemm::gemm_naive_run(data);

    std::cout << "sample2 naive gemm, Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), data.M, data.N);

    CudaTimer::printAll();
    return 0;
}
