#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "utils.h"

// sample
//
//   || ||     || || || ||     || || || ||
//   || ||  X               =  || || || ||
//   || ||     || || || ||     || || || ||


__global__ void gemm_naive(const float* A, const float* B, float* C, int M, int N, int K)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (!(row < M && col < N))
        return;
    
    float sum = 0.0f;
    for (int i = 0; i < K; ++i) {
        int a_cell = row * K + i; //*K שורה
        int b_cell = i * N + col; //*N עמודה
        sum += A[a_cell] * B[b_cell];
    }
    int c_cell = row * N + col;
    C[c_cell] = sum;
    printf("(row,col =(%d,%d)\tC[%d*%d+%d]=%.2f\tC[%d]=%.2f\n", row, col, row, N, col, sum, c_cell, sum);

}

int main()
{
    const int M = 3, K = 2, N = 4;

    std::vector<float> h_A = {
        1, 2,
        3, 4,
        5, 6
    };

    std::vector<float> h_B = {
        1, 2, 3, 4,
        5, 6, 7, 8
    };

    std::vector<float> h_C(M * N, 0);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, M * K * sizeof(float));
    hipMalloc(&d_B, K * N * sizeof(float));
    hipMalloc(&d_C, M * N * sizeof(float));

    hipMemcpy(d_A, h_A.data(), M * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), K * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 block(16, 16);//threads
    dim3 grid((N + 15) / 16, (M + 15) / 16);//blocks

    gemm_naive<<<grid, block>>>(d_A, d_B, d_C, M, N, K);

    hipDeviceSynchronize();


    hipMemcpy(h_C.data(), d_C, M * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);





    std::cout << "Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), M, N);

    
    return 0;
}
