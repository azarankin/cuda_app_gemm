//gemm_simple0.cu cuBlas

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "utils.h"

int main() 
{
    const int M = 3, K = 2, N = 4;

    std::vector<float> h_A {
        1, 2,
        3, 4,
        5, 6
    };  // M x K, Row-Major

    std::vector<float> h_B {
        1, 2, 3, 4,
        5, 6, 7, 8
    };  // K x N, Row-Major

    std::vector<float> h_C(M * N, 0.0f);  // התוצאה

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, h_A.size() * sizeof(float));
    hipMalloc(&d_B, h_B.size() * sizeof(float));
    hipMalloc(&d_C, h_C.size() * sizeof(float));

    hipMemcpy(d_A, h_A.data(), h_A.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), h_B.size() * sizeof(float), hipMemcpyHostToDevice);

    // יצירת handle של cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // הסבר: cuBLAS עובד ב־Column-Major, לכן A ו־B מוחלפים
    // כלומר: C = alpha * B × A + beta * C
    // כדי לקבל C = A × B כמו אצלך, צריך להעביר אופרטור transpose לשניהם

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K,  // cols of C, rows of C, cols of A / rows of B
                &alpha,
                d_B, N,  // B: N×K (ב-CUBLAS: Column-Major)
                d_A, K,  // A: K×M
                &beta,
                d_C, N); // C: N×M

    hipMemcpy(h_C.data(), d_C, h_C.size() * sizeof(float), hipMemcpyDeviceToHost);

    // ניקוי משאבים
    hipblasDestroy(handle);
    hipFree(d_A); hipFree(d_B); hipFree(d_C);

    std::cout << "Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), M, N);

    return 0;
}
