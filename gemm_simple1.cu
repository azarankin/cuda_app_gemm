// gemm_simple1.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "gemm_logic/1_gemm_basic_naive.cuh"
#include "gemm_samples.cuh"
#include "gemm_utils.h"

// sample 1
//
//   || ||     || || || ||     || || || ||
//   || ||  X               =  || || || ||
//   || ||     || || || ||     || || || ||

int main() 
{
    const gemm::Gemm& data = gemm::complicated_sample;


PROFILE_REPEAT(
    gemm::gemm_basic_naive_run(data);
);

    std::vector<float> h_C = gemm::gemm_basic_naive_run(data);

    std::cout << "sample1 basic naive gemm, Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), data.M, data.N);

    CudaTimer::printAll();
    return 0;
}
