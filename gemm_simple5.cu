#include "hip/hip_runtime.h"
// gemm_simple5.cu // stride with  hipMemcpy2D

#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "utils.h"

#define TILE_WIDTH 16

__global__ void gemm_tiled_stride(const float* A, const float* B, float* C,
                                  int M, int N, int K,
                                  int stride_K, int stride_N) {
    __shared__ float tile_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tile_B[TILE_WIDTH][TILE_WIDTH];

    int y = threadIdx.y, x = threadIdx.x;
    int row = blockIdx.y * TILE_WIDTH + y;
    int col = blockIdx.x * TILE_WIDTH + x;

    float sum = 0.0f;
    int num_tiles = (stride_K + TILE_WIDTH - 1) / TILE_WIDTH;

    for (int t = 0; t < num_tiles; ++t) {
        tile_A[y][x] = A[row * stride_K + t * TILE_WIDTH + x];
        tile_B[y][x] = B[(t * TILE_WIDTH + y) * stride_N + col];
        __syncthreads();

        for (int i = 0; i < TILE_WIDTH; ++i) {
            sum += tile_A[y][i] * tile_B[i][x];
        }
        __syncthreads();
    }

    C[row * stride_N + col] = sum;
}

int roundup(int val, int align) 
{
    return ((val + align - 1) / align) * align;
}

int main() {
    const int M = 3, K = 2, N = 4;
    const int stride_K = roundup(K, TILE_WIDTH);
    const int stride_N = roundup(N, TILE_WIDTH);
    const int stride_M = roundup(M, TILE_WIDTH);

    // Compact host arrays

    std::vector<float> h_A = {
        1, 2,
        3, 4,
        5, 6
    };

    std::vector<float> h_B = {
        1, 2, 3, 4,
        5, 6, 7, 8
    };

    // Device pitched allocations (manual stride) using hipMallocPitch not used for simplicity
    float *d_A, *d_B, *d_C;
    size_t pitchA = stride_K * sizeof(float);
    size_t pitchB = stride_N * sizeof(float);
    size_t pitchC = stride_N * sizeof(float);

    hipMalloc(&d_A, stride_K * M * sizeof(float));
    hipMalloc(&d_B, stride_N * K * sizeof(float));
    hipMalloc(&d_C, stride_N * M * sizeof(float));
    hipMemset(d_C, 0, stride_N * M * sizeof(float));

    // Copy compact A to padded device memory efficiently
    hipMemcpy2D(d_A, pitchA, h_A.data(), K * sizeof(float), K * sizeof(float), M, hipMemcpyHostToDevice);
    hipMemcpy2D(d_B, pitchB, h_B.data(), N * sizeof(float), N * sizeof(float), K, hipMemcpyHostToDevice);

    dim3 block(TILE_WIDTH, TILE_WIDTH);
    dim3 grid(stride_N / TILE_WIDTH, stride_M / TILE_WIDTH);
    gemm_tiled_stride<<<grid, block>>>(d_A, d_B, d_C, M, N, K, stride_K, stride_N);
    hipDeviceSynchronize();

    // Copy ROI (M x N) back to compact host array
    std::vector<float> h_C_dense(M * N);
    hipMemcpy2D(h_C_dense.data(), N * sizeof(float),
                 d_C, pitchC, N * sizeof(float), M, hipMemcpyDeviceToHost);

    std::cout << "Matrix C (ROI " << M << "x" << N << "):" << std::endl;
    utils::print_matrix_preview("C", h_C_dense.data(), M, N);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}
