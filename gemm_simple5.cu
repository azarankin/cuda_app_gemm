// gemm_simple5.cu stride with hipMemcpy2D
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include "gemm_logic/5_gemm_cudamemcpy2d.cuh"
#include "gemm_samples.cuh"
#include "utils.h"

//#define TILE_WIDTH 16
// Each || is 16x16     //to refactoring
//
//   || ||     || || || ||     || || || ||
//   || ||  X               =  || || || ||
//   || ||     || || || ||     || || || ||

int main() {
    const gemm::Gemm& data = gemm::basic_sample;

    std::vector<float> h_C = gemm::gemm_cudamemcpy2d_run(data);

    std::cout << "sample5 gemm cudamemcpy2 tiled, Matrix C = A x B:" << std::endl;
    utils::print_matrix_preview("C", h_C.data(), data.M, data.N);

    CudaTimer::printAll();
    return 0;
}
